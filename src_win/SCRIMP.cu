#include "hip/hip_runtime.h"
#include <vector>
#include <numeric>
#include <unordered_map>
#include <float.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reverse.h>
#include <thrust/transform_scan.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include "unistd.h"
#include "SCRIMP.h"
#include "tile.h"

using std::vector;
using std::unordered_map;
using std::make_pair;

namespace SCRIMP {

static const int ISSUED_ALL_DEVICES = -2;



__global__ void cross_correlation_to_ed(float *profile, unsigned int n, unsigned int m) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n) {
        profile[tid] = sqrt(max(2*(1 - profile[tid]), 0.0)) * sqrt((double)m);
    }
}

//This kernel computes a sliding mean with specified window size and a corresponding prefix sum array (A)
template<class DTYPE>
__global__ void sliding_mean(DTYPE* pref_sum,  size_t window, size_t size, DTYPE* means)
{
    const DTYPE coeff = 1.0 / (DTYPE) window;
    size_t a = blockIdx.x * blockDim.x + threadIdx.x;
    size_t b = blockIdx.x * blockDim.x + threadIdx.x + window;

    if(a == 0){
        means[a] = pref_sum[window - 1] * coeff;
    }
    if(a < size - 1){
        means[a + 1] = (pref_sum[b] - pref_sum[a]) * coeff;
    }
}

template<class DTYPE>
__global__ void sliding_norm(DTYPE* cumsumsqr, unsigned int window, unsigned int size, DTYPE* norms) {
    const DTYPE coeff = 1 / (DTYPE) window;
    int a = blockIdx.x * blockDim.x + threadIdx.x;
    int b = blockIdx.x * blockDim.x + threadIdx.x + window;
    if (a == 0) {
        norms[a] = 1 / sqrt(cumsumsqr[window - 1]);
    }
    else if (b < size + window) {
        norms[a] = 1 / sqrt(cumsumsqr[b - 1] - cumsumsqr[a - 1]);
    }
}

template<class DTYPE>
__global__ void sliding_dfdg(const DTYPE *T, const DTYPE *means, DTYPE *df, DTYPE *dg, const int m, const int n) {
    const DTYPE half = 1.0 / (DTYPE) 2.0;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n - 1) {
        df[tid] = (T[tid + m] - T[tid]) * half;
        dg[tid] = (T[tid + m] - means[tid + 1]) + (T[tid] - means[tid]);
    }
}

__global__ void __launch_bounds__(512,4) 
fastinvnorm(double *norm, const double *mean, const double *T, int m, int n) {
   
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int jump = ceil(n / (double) (blockDim.x * gridDim.x));
    int start = jump * tid;
    int end = jump * (tid + 1);
    end = min(end,n);
    if(start >= n) {
        return;
    }
    double sum = 0;
    for(int i = 0; i < m; ++i){ 
        double val = T[i + start] - mean[start];
        sum += val * val;
    }
    norm[start] = sum;
    
    for(int i = start+1; i < end; ++i) {
            norm[i] = norm[i - 1]  + ((T[i-1] - mean[i-1]) + (T[i + m - 1] - mean[i])) * (T[i + m - 1] - T[i - 1]);
    }
    for(int i = start; i < end; ++i) {
        norm[i] = 1.0 / sqrt(norm[i]);
    }
}


void compute_statistics(const double *T, double *norms, double *df, double *dg,
                        double *means, size_t n, size_t m, hipStream_t s, double *scratch)
{
    square<double> sqr;
    dim3 grid(ceil(n / (double) 512), 1,1);
    dim3 block(512, 1, 1);
    
    gpuErrchk(hipPeekAtLastError());
    
    thrust::device_ptr<const double> dev_ptr_T = thrust::device_pointer_cast(T);
    thrust::device_ptr<double> dev_ptr_scratch = thrust::device_pointer_cast(scratch);

    // Compute prefix sum in scratch
    thrust::inclusive_scan(thrust::cuda::par.on(s), dev_ptr_T, dev_ptr_T + n + m - 1,
                           dev_ptr_scratch, thrust::plus<double>());
    gpuErrchk(hipPeekAtLastError());
    // Use prefix sum to compute sliding mean
    sliding_mean<double><<<grid, block, 0, s>>>(scratch, m, n, means);
    gpuErrchk(hipPeekAtLastError());
    
    // Compute differential values
    sliding_dfdg<double><<<grid, block, 0, s>>>(T, means, df,dg,m,n);
    gpuErrchk(hipPeekAtLastError());
    
    // Compute prefix sum of squares in scratch
    thrust::transform_inclusive_scan(thrust::cuda::par.on(s), dev_ptr_T, dev_ptr_T + n + m - 1,
                                     dev_ptr_scratch, sqr,thrust::plus<double>());
    gpuErrchk(hipPeekAtLastError());
    
    // This will be kind of slow on the GPU, may cause latency between tiles
    int workers = n / m + 1;
    fastinvnorm<<<dim3(ceil(workers / (double)512),1,1), dim3(512,1,1), 0, s>>>(norms, means, T, m, n);
    gpuErrchk(hipPeekAtLastError());
    
}

SCRIMPError_t SCRIMP_Operation::init()
{
    for (auto device : devices) {
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        
        T_A_dev.insert(make_pair(device, (double*) 0));
        T_B_dev.insert(make_pair(device, (double*) 0));
        QT_dev.insert(make_pair(device, (double*) 0));
        means_A.insert(make_pair(device, (double*) 0));
        means_B.insert(make_pair(device, (double*) 0));
        norms_A.insert(make_pair(device, (double*) 0));
        norms_B.insert(make_pair(device, (double*) 0));
        df_A.insert(make_pair(device, (double*) 0));
        df_B.insert(make_pair(device, (double*) 0));
        dg_A.insert(make_pair(device, (double*) 0));
        dg_B.insert(make_pair(device, (double*) 0));
        profile_A_dev.insert(make_pair(device,(float*) NULL));
        profile_B_dev.insert(make_pair(device,(float*) NULL));
        profile_A_merged.insert(make_pair(device,(unsigned long long int*) NULL));
        profile_B_merged.insert(make_pair(device,(unsigned long long int*) NULL));
        profile_idx_A_dev.insert(make_pair(device,(unsigned int *) NULL));
        profile_idx_B_dev.insert(make_pair(device,(unsigned int *) NULL));
        scratchpad.insert(make_pair(device, (double*) NULL));

        hipMalloc(&T_A_dev.at(device), sizeof(double) * tile_size);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&T_B_dev.at(device), sizeof(double) * tile_size);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_A_dev.at(device), sizeof(float) * tile_n_x);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_B_dev.at(device), sizeof(float) * tile_n_y);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_idx_A_dev.at(device), sizeof(unsigned int) * tile_n_x);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_idx_B_dev.at(device), sizeof(unsigned int) * tile_n_y);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&QT_dev.at(device), sizeof(double) * tile_n_x);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&means_A.at(device), sizeof(double) * tile_n_x);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&means_B.at(device), sizeof(double) * tile_n_y);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&norms_A.at(device), sizeof(double) * tile_n_x);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&norms_B.at(device), sizeof(double) * tile_n_y);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&df_A.at(device), sizeof(double) * tile_n_x);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&df_B.at(device), sizeof(double) * tile_n_y);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&dg_A.at(device), sizeof(double) * tile_n_x);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&dg_B.at(device), sizeof(double) * tile_n_y);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_A_merged.at(device), sizeof(unsigned long long int) * tile_n_x);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_B_merged.at(device), sizeof(unsigned long long int) * tile_n_y);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&scratchpad.at(device), sizeof(double) * tile_size);
        scratch[device] = new fft_precompute_helper(tile_size, m, true);
        hipEvent_t st, ed, copy;
        hipEventCreate(&ed);
        gpuErrchk(hipPeekAtLastError());
        hipEventCreate(&st);
        gpuErrchk(hipPeekAtLastError());
        hipEventCreate(&copy);
        gpuErrchk(hipPeekAtLastError());
        
        clocks_start.emplace(device, st);
        clocks_end.emplace(device, ed);
        copy_to_host_done.emplace(device, copy);
        hipStream_t s;
        hipStreamCreate(&s);
        gpuErrchk(hipPeekAtLastError());
        streams.emplace(device, s);
    }
    return SCRIMP_NO_ERROR;

}

SCRIMPError_t SCRIMP_Operation::destroy()
{
    for (auto device : devices) {
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        hipFree(T_A_dev[device]);
        hipFree(T_B_dev[device]);
        hipFree(QT_dev[device]);
        hipFree(means_A[device]);
        hipFree(means_B[device]);
        hipFree(norms_A[device]);
        hipFree(norms_B[device]);
        hipFree(df_A[device]);
        hipFree(df_B[device]);
        hipFree(dg_A[device]);
        hipFree(dg_B[device]);
        hipFree(profile_A_dev[device]);
        hipFree(profile_B_dev[device]);
        hipFree(profile_A_merged[device]);
        hipFree(profile_B_merged[device]);
        hipFree(profile_idx_A_dev[device]);
        hipFree(profile_idx_B_dev[device]);
        hipFree(scratchpad.at(device));
        delete scratch[device];
        hipEventDestroy(clocks_start[device]);
        hipEventDestroy(clocks_end[device]);
        hipEventDestroy(copy_to_host_done[device]);
        hipStreamDestroy(streams.at(device));
    }
    return SCRIMP_NO_ERROR;

}

SCRIMPError_t SCRIMP_Operation::do_tile(SCRIMPTileType t, int device, const vector<double> &Ta_h, const vector<double> &Tb_h,
                                        const vector<float> &profile_h, const vector<unsigned int> &profile_idx_h,
                                        const vector<float> &profile_B_h, const vector<unsigned int> &profile_idx_B_h)
{
        size_t start_x = pos_x[device];
        size_t start_y = pos_y[device];
        MPIDXCombine combiner;
        SCRIMPError_t err;
        size_t t_n_x = n_x[device] - m + 1;
        size_t t_n_y = n_y[device] - m + 1;
        printf("tile type = %d start_pos = [%lu, %lu]...\n", t, start_y, start_x);
        hipMemcpyAsync(T_A_dev[device], Ta_h.data() + start_x, sizeof(double) * n_x[device], hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(T_B_dev[device], Tb_h.data() + start_y, sizeof(double) * n_y[device], hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profile_A_dev[device], profile_h.data() + start_x, sizeof(float) * t_n_x, hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profile_idx_A_dev[device], profile_idx_h.data() + start_x, sizeof(unsigned int) * t_n_x, hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        if(self_join) {
            hipMemcpyAsync(profile_B_dev[device], profile_h.data() + start_y, sizeof(float) * t_n_y, hipMemcpyHostToDevice, streams.at(device));
            gpuErrchk(hipPeekAtLastError());
            hipMemcpyAsync(profile_idx_B_dev[device], profile_idx_h.data() + start_y, sizeof(unsigned int) * t_n_y, hipMemcpyHostToDevice, streams.at(device));
            gpuErrchk(hipPeekAtLastError());
        } else if(full_join) {
            hipMemcpyAsync(profile_B_dev[device], profile_B_h.data() + start_y, sizeof(float) * t_n_y, hipMemcpyHostToDevice, streams.at(device));
            gpuErrchk(hipPeekAtLastError());
            printf("start = %lu, size = %lu\n", start_y, profile_idx_B_h.size());
            hipMemcpyAsync(profile_idx_B_dev[device], profile_idx_B_h.data() + start_y, sizeof(unsigned int) * t_n_y, hipMemcpyHostToDevice, streams.at(device));
            gpuErrchk(hipPeekAtLastError());
        }
        // FIXME?: Computing the sliding dot products & statistics for each tile is overkill
        compute_statistics(T_A_dev[device], norms_A[device], df_A[device], dg_A[device], means_A[device], t_n_x, m, streams.at(device), scratchpad[device]);
        gpuErrchk(hipPeekAtLastError());
        compute_statistics(T_B_dev[device], norms_B[device], df_B[device], dg_B[device],  means_B[device], t_n_y, m, streams.at(device), scratchpad[device]);
        gpuErrchk(hipPeekAtLastError());
        thrust::device_ptr<unsigned long long int> ptr_A = thrust::device_pointer_cast(profile_A_merged[device]);
        thrust::transform(thrust::cuda::par.on(streams.at(device)), profile_A_dev[device], profile_A_dev[device] + t_n_x, profile_idx_A_dev[device], profile_A_merged[device], combiner);
        gpuErrchk(hipPeekAtLastError());
        if(self_join || full_join) {
            thrust::device_ptr<unsigned long long int> ptr_B = thrust::device_pointer_cast(profile_B_merged[device]);
            thrust::transform(thrust::cuda::par.on(streams.at(device)), profile_B_dev[device], profile_B_dev[device] + t_n_y, profile_idx_B_dev[device], profile_B_merged[device], combiner);
            gpuErrchk(hipPeekAtLastError());
        }
        SCRIMP_Tile tile(t, T_A_dev[device], T_B_dev[device], df_A[device], df_B[device], dg_A[device], dg_B[device],
                         norms_A[device], norms_B[device], means_A[device], means_B[device],  QT_dev[device],
                         profile_A_merged[device], profile_B_merged[device], start_x, start_y, tile_start_col_position,
                         tile_start_row_position, n_y[device], n_x[device], m, scratch[device], dev_props.at(device), compute_fp64);
        hipEventRecord(clocks_start[device], streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        err = tile.execute(streams.at(device));
        hipEventRecord(clocks_end[device], streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        return err; 

}

void SCRIMP_Operation::get_tile_ordering() {
    tile_ordering.clear();
    size_t num_tile_rows = ceil((size_B - m + 1) / (float) tile_n_y);
    size_t num_tile_cols = ceil((size_A - m + 1) / (float) tile_n_x);

    if(self_join) {
        for(int offset = 0; offset < num_tile_rows - 1; ++offset) {
            for(int diag = 0; diag < num_tile_cols - 1 - offset; ++diag) {
                tile_ordering.emplace_back(diag,diag + offset);
            }
        }

        for(int i = 0; i < num_tile_rows; ++i) {
            tile_ordering.emplace_back(i, num_tile_cols - 1);
        }
    } else {
        // Add upper diagonals one at a time except for edge tiles
        for(int diag = 0; diag < num_tile_cols - 1; ++diag) {
            for(int offset = 0; offset + diag < num_tile_cols - 1 && offset < num_tile_rows - 1; ++offset) {
                    tile_ordering.emplace_back(offset,diag + offset);
            }
        }
        
        // Add lower diagonals one at a time except for edge tiles
        for(int diag = 1; diag < num_tile_rows - 1; ++diag) {
            for(int offset = 0; offset + diag < num_tile_rows - 1 && offset < num_tile_cols - 1; ++offset) {
                tile_ordering.emplace_back(offset+diag,offset); 
            }
        }
    
        // Add the corner edge tile
        tile_ordering.emplace_back(num_tile_rows - 1, num_tile_cols - 1);

        int x = 0;
        int y = 0;

        // Alternate between adding final row and final column edge tiles
        while (x < num_tile_cols - 1 && y < num_tile_rows - 1) {
            tile_ordering.emplace_back(y, num_tile_cols - 1);
            tile_ordering.emplace_back(num_tile_rows - 1, x);
            ++x;
            ++y;
        }

        // Add any remaining final row edge tiles
        while (x < num_tile_cols - 1){
            tile_ordering.emplace_back(num_tile_rows - 1,x);
            ++x;
        }
    
        // Add any remaining final column edge tiles
        while (y < num_tile_rows - 1) {
            tile_ordering.emplace_back(y, num_tile_cols - 1);
            ++y;
        }
        
    }
    total_tiles = tile_ordering.size();
}


bool SCRIMP_Operation::pick_and_start_next_tile(int dev, const vector<double> &Ta_h, const vector<double> &Tb_h,
                                                const vector<float> &profile_h, const vector<unsigned int> &profile_idx_h,
                                                const vector<float> &profile_B_h, const vector<unsigned int> &profile_idx_B_h)
{
    
    bool done = false;
    int tile_row = tile_ordering.front().first;
    int tile_col = tile_ordering.front().second;
    pos_x[dev] = tile_col * tile_n_x;
    pos_y[dev] = tile_row * tile_n_y;
    n_x[dev] = min(tile_size, size_A - pos_x[dev]);
    n_y[dev] = min(tile_size, size_B - pos_y[dev]);
    if(self_join) {
        if(tile_row == tile_col) {
            //partial tile on diagonal
            do_tile(SELF_JOIN_UPPER_TRIANGULAR, dev, Ta_h, Tb_h, profile_h, profile_idx_h, profile_B_h, profile_idx_B_h);
        } else {
            // full tile
            do_tile(SELF_JOIN_FULL_TILE, dev, Ta_h, Tb_h, profile_h, profile_idx_h, profile_B_h, profile_idx_B_h);
        }
    } else if(full_join) {
        do_tile(AB_FULL_JOIN_FULL_TILE, dev, Ta_h, Tb_h, profile_h, profile_idx_h, profile_B_h, profile_idx_B_h);
    } else {
        do_tile(AB_JOIN_FULL_TILE, dev, Ta_h, Tb_h, profile_h, profile_idx_h, profile_B_h, profile_idx_B_h);
    }
    tile_ordering.pop_front();
    if(tile_ordering.empty()){
        done = true;
    }
    return done;
}


void merge_partial_on_host(vector<unsigned long long int> &profile_to_merge, vector<float> &merge_target, vector<unsigned int> &merge_idx_target, size_t merge_start, size_t tile_sz)
{
    auto iter_begin = thrust::make_zip_iterator(thrust::make_tuple(merge_target.data() + merge_start, merge_idx_target.data() + merge_start, profile_to_merge.data()));
    auto iter_end = thrust::make_zip_iterator(thrust::make_tuple(merge_target.data() + merge_start + tile_sz, merge_idx_target.data() + merge_start + tile_sz, profile_to_merge.data() + tile_sz));
    thrust::for_each(iter_begin, iter_end, max_with_index());

}


int SCRIMP_Operation::issue_and_merge_tiles_on_devices(const vector<double> &Ta_host, const vector<double> &Tb_host,
                                                       vector<float> &profile_A_full_host, vector<unsigned int> &profile_idx_A_full_host,
                                                       vector<float> &profile_B_full_host, vector<unsigned int> &profile_idx_B_full_host,
                                                       vector<vector<unsigned long long int>> &profileA_h,
                                                       vector<vector<unsigned long long int>> &profileB_h,
                                                       int last_device_idx = ISSUED_ALL_DEVICES)
{
    bool done = last_device_idx != ISSUED_ALL_DEVICES;
    int last_dev = ISSUED_ALL_DEVICES; 
    if(last_device_idx == ISSUED_ALL_DEVICES) {
        last_device_idx = devices.size() - 1;
    }
    for(int i = 0; i <= last_device_idx; ++i) {
        int device = devices.at(i);
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profileA_h.at(i).data(), profile_A_merged[device], sizeof(unsigned long long int) * (n_x[device] - m + 1), hipMemcpyDeviceToHost, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        if(self_join || full_join) {
            hipMemcpyAsync(profileB_h.at(i).data(), profile_B_merged[device], sizeof(unsigned long long int) * (n_y[device] - m + 1), hipMemcpyDeviceToHost, streams.at(device));
            gpuErrchk(hipPeekAtLastError());
        }
        hipEventRecord(copy_to_host_done[device], streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        n_x_2[device] = n_x[device];
        n_y_2[device] = n_y[device];
        pos_x_2[device] = pos_x[device];
        pos_y_2[device] = pos_y[device];
        if(!done) {
            done = pick_and_start_next_tile(device, Ta_host, Tb_host, profile_A_full_host, profile_idx_A_full_host, profile_B_full_host, profile_idx_B_full_host);
            if(done) {
                last_dev = i;
            }
        }
    }

    for(int i = 0; i <= last_device_idx; ++i) {
        int device = devices.at(i);
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        hipEventSynchronize(copy_to_host_done[device]);
        gpuErrchk(hipPeekAtLastError());
        merge_partial_on_host(profileA_h.at(i), profile_A_full_host, profile_idx_A_full_host, pos_x_2[device], (n_x_2[device] - m + 1));
        gpuErrchk(hipPeekAtLastError());
        if(self_join) {
            merge_partial_on_host(profileB_h.at(i), profile_A_full_host, profile_idx_A_full_host, pos_y_2[device], (n_y_2[device] - m + 1));
            gpuErrchk(hipPeekAtLastError());
        } else if(full_join) {
            merge_partial_on_host(profileB_h.at(i), profile_B_full_host, profile_idx_B_full_host, pos_y_2[device], (n_y_2[device] - m + 1));
            gpuErrchk(hipPeekAtLastError());
        }
        completed_tiles++;
        printf("%f percent complete\n", (completed_tiles / (float) total_tiles) * 100);
    }
    return last_dev;
}


SCRIMPError_t SCRIMP_Operation::do_join(const vector<double> &Ta_host, const vector<double> &Tb_host, vector<float> &profile, vector<unsigned int> &profile_idx, vector<float> &profile_B, vector<unsigned int> &profile_idx_B)
{

    vector< vector<unsigned long long int> > profileA_h(devices.size(), vector<unsigned long long int>(tile_n_y)), profileB_h(devices.size(), vector<unsigned long long int>(tile_n_x));
    bool done = false;
    int last_dev = ISSUED_ALL_DEVICES;
    get_tile_ordering();
    printf("Performing join with %lu tiles.\n", tile_ordering.size() );
    for(int i = 0; i < devices.size(); ++i) {
        int device = devices.at(i);
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        done = pick_and_start_next_tile(device, Ta_host, Tb_host, profile, profile_idx, profile_B, profile_idx_B);
        gpuErrchk(hipPeekAtLastError());
        if (done) {
            last_dev = i;
            break;
        }
    }

    while(last_dev == ISSUED_ALL_DEVICES) {
        last_dev = issue_and_merge_tiles_on_devices(Ta_host, Tb_host, profile, profile_idx, profile_B, profile_idx_B, profileA_h, profileB_h);
    }

    issue_and_merge_tiles_on_devices(Ta_host, Tb_host, profile, profile_idx, profile_B, profile_idx_B, profileA_h, profileB_h, last_dev);

    return SCRIMP_NO_ERROR;
}

void do_SCRIMP(const vector<double> &Ta_h, const vector<double> &Tb_h, vector<float> &profile_h, vector<unsigned int> &profile_idx_h,
               vector<float> &profile_B_h, vector<unsigned int> &profile_idx_B_h, const unsigned int m, const size_t max_tile_size, 
               const vector<int> &devices, bool self_join, bool fp64, bool full_join, size_t start_row, size_t start_col)
{
    if(devices.empty()) {
        printf("Error: no gpu provided\n");
        exit(0);
    }
    // Allocate and initialize memory
    clock_t start, end;
    SCRIMP_Operation op(Ta_h.size(), Tb_h.size(), m, max_tile_size, devices, self_join, fp64, full_join, start_row, start_col);
    op.init();
    gpuErrchk(hipPeekAtLastError());
    start = clock();
    if(self_join) {
        op.do_join(Ta_h, Ta_h, profile_h, profile_idx_h, profile_B_h, profile_idx_B_h);
    } else {
        op.do_join(Ta_h, Tb_h, profile_h, profile_idx_h, profile_B_h, profile_idx_B_h);

    }
    hipDeviceSynchronize();
    end = clock();
    gpuErrchk(hipPeekAtLastError());
    op.destroy();
    gpuErrchk(hipPeekAtLastError());
    printf("Finished SCRIMP to generate partial matrix profile of size %lu in %f seconds on %lu devices:\n", profile_h.size(), (end - start) / (double) CLOCKS_PER_SEC, devices.size());
}

//Reads input time series from file
template<class DTYPE>
void readFile(const char* filename, vector<DTYPE>& v, const char *format_str) 
{
    FILE* f = fopen( filename, "r");
    if(f == NULL){
        printf("Unable to open %s for reading, please make sure it exists\n", filename);
        exit(0);
    }
    DTYPE num;
    while(!feof(f)){
            fscanf(f, format_str, &num);
            v.push_back(num);
        }
    v.pop_back();
    fclose(f);
}
    
}

int main(int argc, char** argv) {


    bool fp_64 = false;
    bool full_join = false;
    bool self_join = true;
    size_t start_row = 0;
    size_t start_col = 0;
    int max_tile_size = (1 << 20);
    int opt;
    vector<int> devices;
    vector<double> Ta_h, Tb_h;
    char *output_B_prefix, *input_B;
    while ((opt = getopt(argc, argv, "df:r:c:s:b:g:")) != -1) {
        switch (opt) {
        case 'd': fp_64 = true; break;
        case 'f': output_B_prefix = optarg; full_join = true; break;
        case 'r': start_row = atoi(optarg); break;
        case 'c': start_col = atoi(optarg); break;
        case 's': max_tile_size = atoi(optarg); break;
        case 'b': input_B = optarg; self_join = false; break;
        case 'g': devices.push_back(atoi(optarg)); break;
        default:
            exit(EXIT_FAILURE);
        }
    }
    if(self_join && full_join) {
        printf("error: invalid argument combination -f flag can only be used in ab-joins");
        exit(EXIT_FAILURE);
    }

    int index = optind;
    int window_size = atoi(argv[index++]);
    char *input_A = argv[index++];
     
    SCRIMP::readFile<double>(input_A, Ta_h, "%lf");
    
    if(!self_join) {
        SCRIMP::readFile<double>(input_B, Tb_h, "%lf"); 
    }

    int n_x = Ta_h.size() - window_size + 1;
    int n_y;
    if(self_join) {
       n_y = n_x;
    } else {
       n_y = Tb_h.size() - window_size + 1;
    }

    vector<float> profile(n_x, CC_MIN);
    vector<unsigned int> profile_idx(n_x, 0);
    vector<float> profile_B;
    vector<unsigned int> profile_idx_B;

    if(full_join){
        profile_B = vector<float>(n_y, CC_MIN);
        profile_idx_B = vector<unsigned int>(n_y, 0);
    }

    hipFree(0);
    
    if(devices.empty()) {
        // Use all available devices
        printf("using all devices\n"); 
        int num_dev;
        hipGetDeviceCount(&num_dev);
        for(int i = 0; i < num_dev; ++i){ 
            devices.push_back(i);
        }
    }

    printf("Starting SCRIMP\n");     
    SCRIMP::do_SCRIMP(Ta_h, Tb_h, profile, profile_idx, profile_B, profile_idx_B, window_size, max_tile_size, devices, self_join, fp_64, full_join, start_row, start_col);
    
    printf("Now writing result to files\n");
    FILE* f1 = fopen( argv[index++], "w");
    FILE* f2 = fopen( argv[index++], "w");
    FILE* f3, *f4;
    for(int i = 0; i < profile.size(); ++i){
         fprintf(f1, "%f\n", sqrt(max(2*window_size*(1 - profile[i]), 0.0)));
         fprintf(f2, "%u\n", profile_idx[i] + 1);
    }
    fclose(f1);
    fclose(f2);
    if(full_join) {
        f3 = fopen(strcat(output_B_prefix,"_mp") , "w");
        f4 = fopen(strcat(output_B_prefix,"i"), "w");
        for(int i = 0; i < profile_B.size(); ++i) {
            fprintf(f3, "%f\n", sqrt(max(2*window_size*(1 - profile_B[i]), 0.0)));
            fprintf(f4, "%u\n", profile_idx_B[i] + 1);
        }
    }
    if(full_join) {
        fclose(f3);
        fclose(f4);
    }
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipDeviceReset());
    printf("Done\n");
    return 0;
}

